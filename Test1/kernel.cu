﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

#define SIZE 8192

using namespace std;
using namespace std::chrono;






int main()
{
    //CUDA Runtime API
    int DeviceCount;
    hipGetDeviceCount(&DeviceCount);
    cout << "Number of Device : " << DeviceCount << endl;

    hipDeviceProp_t prop;    //structure that stores device properties
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice name : %s", prop.name);
    printf("\nMultiprocessor Count : %d ", prop.multiProcessorCount);
    printf("\nMemory Clock Rate (KHz) : %d", prop.memoryClockRate/(1000*1000));
    printf("\nMemory Pool Supported : %d", prop.memoryPoolsSupported);
    printf("\nTotal Global Memory : %d MB", prop.totalGlobalMem/(1024*1024));

    return 0;
}
