#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

#define SIZE 8192

using namespace std;
using namespace std::chrono;

__global__ void hello() {
    
    int warp_ID = threadIdx.x / warpSize;
    
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("\nGlobal thread ID : %d -- Warp ID : %d", global_thread_id, warp_ID);
    
}

    //printf("\nThe block ID is %d --- The thread ID is %d --- The warp ID %d", blockIdx.x, threadIdx.x,warp_ID );

__global__ void vectorAdd(int* A, int* B, int* C, int n)
{
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int i = global_thread_id;
    C[i] = A[i] + B[i];
}





int main()
{
    /*hello << <2, 64 >> > ();
    hipDeviceSynchronize();*/

    //hello << <2, 512 >> > ();
    const int vector_size = 8192;

    int vect1[vector_size];
    int vect2[vector_size];
    int out[vector_size];
    for (int i = 0; i < vector_size; i++)
    {
        vect1[i] = i * 2 + 3;
    }
    cout << "\nSuccessfully initialized vector 1" << endl;
    for (int j = 0; j < vector_size; j++)
    {
        vect2[j] = j * 3 + 2;
    }
    cout << "\nSuccessfully initialized vector 2" << endl;
    //
    //
    ////function of addition in CPU
    auto start1 = high_resolution_clock::now();
    for (int k = 0; k < vector_size; k++)
    {
        out[k] = vect1[k] + vect2[k];
    }
    cout << "Addtion completed !" << endl;
    auto stop1 = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop1 - start1);

    cout << "\nDuration for CPU is : " << duration.count() << endl;


    int* A, * B, * C; 
    int* d_A, * d_B, * d_C;
    int size = SIZE * sizeof(int);

    A = (int*)malloc(size);
    B = (int*)malloc(size);
    C = (int*)malloc(size);

    //allocate memory in GPU
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);


    //initialize the inputs
    for (int i = 0; i < SIZE; i++)
    {
        A[i] = i;
        B[i] = SIZE - i;
    }

    //Create Events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //copy host memory to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    hipEventRecord(start);
    vectorAdd << <8, 1024 >> > (d_A, d_B, d_C, SIZE);
    hipEventRecord(stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time : %f milliseconds\n", milliseconds);
   

    printf("\nExecution Finished !");
    /*for (int i = 0; i < SIZE; i++)
    {
        printf("%d + %d = %d", A[i], B[i], C[i]);
        printf("\n");
    }*/

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);










    return 0;
}
