#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void hello() {
    
    // print the block and thread IDs
    printf("\nThe block ID is %d --- The thread ID is %d", blockIdx.x, threadIdx.x);
}


int main()
{
    hello << <2, 8 >> > ();

    return 0;
}
